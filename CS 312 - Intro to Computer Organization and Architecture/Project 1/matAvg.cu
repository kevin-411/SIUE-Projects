#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <fstream>
#include <string>
using namespace std;


/**************************************************************************
 * **************************************************************************
 *					error  checking stufff
 ***************************************************************************
 ***************************************************************************/ 
// Enable this for error checking
#define CUDA_CHECK_ERROR

#define CudaSafeCall( err )     __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()        __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
	#ifdef CUDA_CHECK_ERROR
	
	#pragma warning( push )
	#pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
	
	do
	{
		if ( hipSuccess != err )
		{
			fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
					 file, line, hipGetErrorString( err ) );
			exit( -1 );
		}
	} while ( 0 );
	
	#pragma warning( pop )
	
	#endif  // CUDA_CHECK_ERROR
	
	return;
}

inline void __cudaCheckError( const char *file, const int line )
{
	#ifdef CUDA_CHECK_ERROR
	
	#pragma warning( push )
	#pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
	
	do
	{
		hipError_t err = hipGetLastError();
		if ( hipSuccess != err )
		{
			fprintf( stderr, "cudaCheckError() failed at %s:%i : %s.\n",
					 file, line, hipGetErrorString( err ) );
			exit( -1 );
		}
		
		// More careful checking. However, this will affect performance.
		// Comment if not needed.
		err = hipDeviceSynchronize();
		if( hipSuccess != err )
		{
			fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s.\n",
					 file, line, hipGetErrorString( err ) );
			exit( -1 );
		}
	} while ( 0 );
	
	#pragma warning( pop )
	
	#endif // CUDA_CHECK_ERROR
	
	return;
}

/**************************************************************************
 * *************************************************************************
 *					end of error checking stuff
 ***************************************************************************
 **************************************************************************/ 



// function takes an array pointer, and the number of rows and cols in the array, and 
// allocates and intializes the array to a bunch of random numbers
// Note that this function creates a 1D array that is a flattened 2D array
// to access data item data[i][j], you must can use data[(i*rows) + j]
void makeRandArray( unsigned int *& data, unsigned int rows, unsigned int cols, unsigned int seed )
{
	// allocate the array
	data = new unsigned int[ rows*cols ];
	
	// seed the number generator
	// you should change the seed to get different values
	srand( seed );
	
	// populate the array
	
	for( unsigned int i = 0; i < rows*cols; i++ )
	{
		data[i] = rand() % 10000 + 1; // number between 1 and 10000
		//cerr << data[i] << " ";
	}
	//cerr << endl;
	
}

//*******************************//
// your kernel here!!!!!!!!!!!!!!!!!
//*******************************//
__global__ void matavgKernel(int* mAvg, int rows, int cols, int* dNums)
{
	int rowID=threadidx.y + (blockIDx.y * blockDim.y);
	int colID=threadidx.x + (blockIDx.x * blockDim.x);

	int rowMin=((rowID>0)?rowID-1:0);
	int rowMax=((colID<Rows-1?rowID+1:0);
	int colMin=((colID>0)?colID-1:0);
	int colMax=((rowID<Cols-1?colID+1:0);
	float avg=0;
	int count=0, sum=0;

	for(i=rowMin;i<rowMax;i++){
	 for(j=colMin;j<colMax;j++){
	  count++;
	  sum += dNums[(i*Cols)+j];
	 }
	}
	int Atmp = *((int*)(&avg));
	atomicMax(mAvg,Atmp);
}


int main( int argc, char* argv[] ) 
{
	if(  argc < 3 || argc > 4  )
	{		
		cerr<<"usage: exe [num rows] [num cols] [seed value (optional)]" << endl;
		exit( -1 );
	}
	
	unsigned int rows, cols, seed, rowIDOfMaxCell = 0, colIDOfMaxCell = 0;
	unsigned int *host_data;
	unsigned int dataSize;
	float maxCellAvg = 0;
	int maxCellAvgAsInt = 0;
	{
		stringstream ss1;
		ss1 << argv[1];
		ss1 >> rows;
	}
	{
		stringstream ss1;
		ss1 << argv[2];
		ss1 >> cols;
	}
	{
		if( argc < 3 )
		{
			seed = 1;
		}
		else
		{
			stringstream ss1;
			ss1 << argv[3];
			ss1 >> seed;
		}	
	}
	makeRandArray( host_data, rows, cols, seed );
	dataSize = rows*cols;
	
	/***********************************
	 *	 create a cuda timer to time execution
	 **********************************/
	hipEvent_t startTotal, stopTotal;
	float timeTotal;
	hipEventCreate(&startTotal);
	hipEventCreate(&stopTotal);
	hipEventRecord( startTotal, 0 );
	/***********************************
	 *	 end of cuda timer creation
	 **********************************/
	
	
	/////////////////////////////////////////////////////////////////////
	///////////////////////  YOUR CODE HERE       ///////////////////////
	/////////////////////////////////////////////////////////////////////
	/*
	 *	 You need to implement your kernel as a function at the top of this file.
	 *	 Here you must 
	 *	 1) allocate device memory
	 *	 2) set up the grid and block sizes
	 *	 3) call your kenrnel
	 *	 4) get the result back from the GPU
	 *	 
	 *	 
	 *	 to use the error checking code, wrap any  cudamalloc functions as follows:
	 *		CudaSafeCall( hipMalloc( &pointer_to_a_device_pointer, length_of_array * sizeof( int ) ) );
	 *	 Also, place the following function call immediately after you call your kernel
	 *	 ( or after any other cuda call that you think might be causing an error )
	 *		CudaCheckError();
	 */

	 unsigned int rows, cols, seed, rowIDOfMaxCell = 0, colIDOfMaxCell = 0;
	unsigned int *host_data;
	unsigned int dataSize;
	float maxCellAvg = 0;
	int maxCellAvgAsInt = 0;

	dim3 threadsperblock(16,32);
	dim3 numblock((cols + threadsperblock.x-1)/threadsperblock.x, (rows + threadsperblock.y-1)/threadsperblock.y));
	dataSize = rows * cols;
	host_data = new(datasize);
	int* Dmatrix;

	cudasafecall(cudamalloc(Dmatrix,dataSize * sizeof(int));
	cudamemcpy(Dmatrix,host_data,dataSize*,sizeof(int));
	cudamemcpy.hosttodevice;

	MAKernal<<<numblock, treadsperblock>>>(Mavg,rows,cols,Dmatrix);
	
	
	
	/***********************************
	 *	 Stop and destroy the cuda timer 
	 **********************************/
	hipEventRecord( stopTotal, 0 );
	hipEventSynchronize( stopTotal );
	hipEventElapsedTime( &timeTotal, startTotal, stopTotal );
	hipEventDestroy( startTotal );
	hipEventDestroy( stopTotal );
	/***********************************
	 *	 end of cuda timer destruction
	 **********************************/
	
	std::cerr << "Total time in seconds: " << timeTotal / 1000.0 << std::endl;
	std::cerr << "Max cell neighborhood avg: " << maxCellAvg  <<std::endl;
	std::cerr << "Max cell neighborhood address: ("
	<< rowIDOfMaxCell << ", " << colIDOfMaxCell << ")" <<std::endl;
	
}


